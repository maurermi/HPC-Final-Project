#include "hip/hip_runtime.h"
#include <stdio.h>
#include <chrono>
#include <ctime>
#include <string>
#include "SHA256.h"

#define DIFFICULTY 0xff000000
#define NUMTHREAD 1024

// time program
double CLOCK() {
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC, &t);
	return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}

// determine if the hash is < difficulty
bool checkVal(uint8_t* hash) {
	uint32_t diff = DIFFICULTY;
	// cast difficulty to be a string of uint8_t's like the SHA256 library does
	uint8_t * d = static_cast<uint8_t*>(static_cast<void*>(&diff));

	for(int i = 0; i < 8; i++) {
		// compare each uint8_t within the hash to those in difficulty
		// if hash is ever smaller, this hash wins
		// if hash is ever larger, it loses (if they are the same, it loses)
		if(*(d+i) > *(hash + i)) {
			return true;
		}
		else if(*(d+i) < *(hash+i)) {
			return false;
		}
	}
	return false;
}

__global__ void compute_hash(uint32_t start_value, uint8_t** hashes) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	SHA256 sha;
	sha.update(reinterpret_cast<char*>(start_value + i));
	hashes[i] = sha.digest();
}

int main(int argc, char ** argv) {

	double start, finish;
	uint8_t * digest;
	bool solved = false;
	int start_num = 0;
	uint64_t val[1];
	uint8_t **hashes; hipMallocManaged(&hashes, NUMTHREAD*sizeof(*uint8_t));
	start = CLOCK();
	// continue until hash is found
	do {
		compute_hash<<<1, NUMTHREAD>>>(0, hashes);
		for (int i = start_num; i < NUMTHREAD; i++) {
			digest = hashes[i];
			if (checkVal(digest)) {
				*val = i;
				break;
			}
		}
		start_num += NUMTHREAD;
	} while(!solved);
	finish = CLOCK();

	printf("Block solved in %f ms\n", finish-start);
	//printf("%d attempts\n", (*val) - 1);
	printf("%s\n", SHA256::toString(digest).c_str());

	//delete[] digest;
	hipFree(hashes);
	
	return EXIT_SUCCESS;
}
